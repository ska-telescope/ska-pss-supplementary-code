// make
// then example call: ./cufft_loop > cufft_loop_out.csv &
// uses in-place data calc

// ND = normal distribution
// uses sine signal+ND-noise in real part of the complex input vector, just ND-noise for imaginary part
// FFTs with N equal to number(s) in Narray
// Forward and Inverse FFTs, carried out numruns times, each time with new input data
// output performance time (includes CUDA memcopy times)
// plan setup is outside of the loop 

#include <stdlib.h>
#include <math.h>
#include <complex.h>
#include <iostream>
#include <chrono>
#include <random>

// following
//https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuFFT/1d_c2c/1d_c2c_example.cpp
//https://docs.nvidia.com/cuda/cufft/index.html#function-cufftexecc2c-cufftexecz2z
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime.h>

//#include <cufft_utils.h>
//------- not found on sharabha

// ----- how many tests with the same plan?
auto numruns=10000;

int Narray[]={1024*1024*2,1024*1024*4,1024*1024*8};
//int Narray[]={1024*1024*8,1024*1024*4,1024*1024*2};
//int Narray[]={1024*1024*8,1024*1024*8,1024*1024*2,1024*1024*4};
//int Narray[]={16,32,64};
//int Narray[]={1024*1024*2};

// test performance for non-multiple of 2 
//int Narray[]={1024*1024*3,1024*1024*5,1024*1024*2+1024*500};

// do half-hour run and 1Mio run
//int Narray[]={1024*1024*8*4,1024*1024*8*2,1024*1024};


#define tscan 10.0*60.0 

int main()
{
    std::random_device rd;
    std::mt19937 rng(rd());
    std::normal_distribution<float> generate_data(128,30);
    std::normal_distribution<float> generate_amplitude(30,5 );
    std::normal_distribution<float> generate_frequency(1000,100);
// csv header
    std::cout<<"Hcufft,HSINpND,Nlauf,Direction,runum,time,unitns\n";

 // ----- how many of the Ns should be done?
    for (int ni {0}; ni<3; ++ni)
    {
        int Nlauf=Narray[ni];    
// handles for plan and (default) stream        
        hipfftHandle plan;
        hipStream_t stream = NULL;

        using scalar_type = float;
        using data_type = std::complex<scalar_type>;


// --- define data array [CPU]
        std::vector<data_type> input_data(Nlauf);

// --- define data array pointer (GPU)
        hipfftComplex *d_data = nullptr;

// cuda has only one plan, not estimate/measure
        int batchnr =1;   // maybe to be changed later [combining several DM-data sets here could help with speed]
        hipfftCreate(&plan);
        hipfftPlan1d(&plan, Nlauf, HIPFFT_C2C, batchnr);

// Create device data arrays --- this should only be done once (cudafree also only done once)
        hipMalloc(reinterpret_cast<void **>(&d_data), sizeof(data_type) * input_data.size());


// ============================= FORWARD
//------- do for the same N (same plan), numruns times
        for(int k=0; k<numruns; k++)
        {
            double freq=generate_frequency(rng);
            double amp=generate_amplitude(rng);
            for(int i=0; i<Nlauf; i++)
            {
                double t = static_cast<double>(i)* tscan / static_cast<double>(Nlauf);
                double pulse_i=amp * std::sin(2.0 * M_PI * freq * t); 
    //            input_data[i][0] = pulse_i + generate_data(rng);
    //            input_data[i][1] = generate_data(rng);
                input_data[i] = data_type(pulse_i+generate_data(rng), generate_data(rng));
            }


//--------- start stop watch here
            auto fft_start = std::chrono::high_resolution_clock::now();
 // copy input_data (CPU) to d_data (GPU)
            hipMemcpy(d_data, input_data.data(), sizeof(data_type) * input_data.size(), hipMemcpyHostToDevice);

//       Identical pointers to data and output arrays implies in-place transformation
            hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);

//CUDA kernels are asynchronous, 
//forces the program to wait for all previously issued commands in all streams on the device to finish before continuing
// can slow down program
            hipDeviceSynchronize();

// copy.modified. d_data (GPU) to input_data (CPU) -> overwrite
            hipMemcpy(input_data.data(), d_data, sizeof(data_type) * input_data.size(), hipMemcpyDeviceToHost);
 
            auto fft_stop = std::chrono::high_resolution_clock::now();
//--------- end stop watch
//        std::cout<<"Time elapsed: "<<std::chrono::duration_cast<std::chrono::nanoseconds>(fft_stop - fft_start).count()<<",ns\n";
//        std::printf("=====\n");
            std::cout<<"cufft,SINpND,"<<Nlauf<<",FORWARD,"<<k<<","<<std::chrono::duration_cast<std::chrono::nanoseconds>(fft_stop - fft_start).count()<<",ns\n";
        }



// ========================= Backward
//------- do for the same N (same plan), numruns times
        for(int k=0; k<numruns; k++)
        {
            double freq=generate_frequency(rng);
            double amp=generate_amplitude(rng);
            for(int i=0; i<Nlauf; i++)
            {
                double t = static_cast<double>(i)* tscan / static_cast<double>(Nlauf);
                double pulse_i=amp * std::sin(2.0 * M_PI * freq * t); 
    //            input_data[i][0] = pulse_i + generate_data(rng);
    //            input_data[i][1] = generate_data(rng);
                input_data[i] = data_type(pulse_i+generate_data(rng), generate_data(rng));
            }

//--------- start stop watch here
            auto fft_start = std::chrono::high_resolution_clock::now();
//            cudaDeviceSynchronize();   //---- added to make comparable measurement, remove to speed up things later
 // copy input_data (CPU) to d_data (GPU)
            hipMemcpy(d_data, input_data.data(), sizeof(data_type) * input_data.size(), hipMemcpyHostToDevice);

//     *  Identical pointers to data and output arrays implies in-place transformation
            hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);

//CUDA kernels are asynchronous, 
//forces the program to wait for all previously issued commands in all streams on the device to finish before continuing
// can slow down program
            hipDeviceSynchronize();

// copy.modified. d_data (GPU) to input_data (CPU) -> overwrite
            hipMemcpy(input_data.data(), d_data, sizeof(data_type) * input_data.size(), hipMemcpyDeviceToHost);

            auto fft_stop = std::chrono::high_resolution_clock::now();
//--------- end stop watch
//        std::cout<<"Time elapsed: "<<std::chrono::duration_cast<std::chrono::nanoseconds>(fft_stop - fft_start).count()<<",ns\n";
//        std::printf("=====\n");
            std::cout<<"cufft,SINpND,"<<Nlauf<<",BACKWARD,"<<k<<","<<std::chrono::duration_cast<std::chrono::nanoseconds>(fft_stop - fft_start).count()<<",ns\n";
        }




        /* free resources */
//       cudaDeviceSynchronize();   //---- added to make comparable measurement, remove to speed up things later
        hipFree(d_data);

        hipfftDestroy(plan);

        hipStreamDestroy(stream);

        hipDeviceReset();
    }
    return 0;
}
