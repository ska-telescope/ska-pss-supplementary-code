//  make
// example call: ./cusanity > inout.csv &
// uses in-place data calc
//
// uses sine signal+ND-noise in real part of the complex input vector, just ND-noise for imaginary part
// writes out input sine wave, input array (real+ima) and then output array (real, ima) of each run 
// for double checking with other FFT routines and recovering test of input sine signal


#include <stdlib.h>
#include <math.h>
#include <complex.h>
#include <iostream>
#include <chrono>
#include <random>

// following
//https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuFFT/1d_c2c/1d_c2c_example.cpp
//https://docs.nvidia.com/cuda/cufft/index.html#function-cufftexecc2c-cufftexecz2z
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime.h>

//#include <cufft_utils.h>
//------- not found on sharabha

// ----- how many tests with the same plan?
auto numruns=2;
int Narray[]={1024*1024*2};


#define tscan 10.0*60.0 

int main()
{
    std::random_device rd;
    std::mt19937 rng(rd());
    std::normal_distribution<float> generate_data(128,10 );
    std::normal_distribution<float> generate_amplitude(30,5 );
    std::normal_distribution<float> generate_frequency(1000,10 );

    std::cout<<"cuFFTWwhat,amp,freq,runid,idx,t_freq,real,ima\n";

 // ----- how many of the Ns should be done?
    for (int ni {0}; ni<1; ++ni)
    {
        int Nlauf=Narray[ni];    
// handles for plan and (default) stream        
        hipfftHandle plan;
        hipStream_t stream = NULL;

        using scalar_type = float;
        using data_type = std::complex<scalar_type>;


// --- define data array [CPU]
        std::vector<data_type> input_data(Nlauf);

// --- define data array pointer (GPU)
        hipfftComplex *d_data = nullptr;

// cuda has only one plan, not estimate/measure
        int batchnr =1;   // maybe to be changed later [combining several DM-data sets here could help with speed]
        hipfftCreate(&plan);
        hipfftPlan1d(&plan, Nlauf, HIPFFT_C2C, batchnr);

// Create device data arrays --- this should only be done once (cudafree also only done once)
        hipMalloc(reinterpret_cast<void **>(&d_data), sizeof(data_type) * input_data.size());


// ============================= FORWARD
//------- do for the same N (same plan), numruns times
        for(int k=0; k<numruns; k++)
        {
           double freq=generate_frequency(rng);
           double amp=generate_amplitude(rng);
           for(int i=0; i<Nlauf; i++)
            {
                double t = static_cast<double>(i)* tscan / static_cast<double>(Nlauf);
                double pulse_i=amp * std::sin(2.0 * M_PI * freq * t); 
    //            input_data[i][0] = pulse_i + generate_data(rng);
    //            input_data[i][1] = generate_data(rng);
                input_data[i] = data_type(pulse_i+generate_data(rng), generate_data(rng));
                std::cout<<"cuFFTWin,"<<amp<<","<<freq<<","<<k<<","<<i<<","<<t<<","<<input_data[i].real()<<","<<input_data[i].imag()<<"\n";
            }


//--------- start stop watch here
            auto fft_start = std::chrono::high_resolution_clock::now();
 // copy input_data (CPU) to d_data (GPU)
            hipMemcpy(d_data, input_data.data(), sizeof(data_type) * input_data.size(), hipMemcpyHostToDevice);

    /*
     * Note:
     *  Identical pointers to data and output arrays implies in-place transformation
     */
            hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
 
//CUDA kernels are asynchronous, 
//forces the program to wait for all previously issued commands in all streams on the device to finish before continuing
// can slow down program
            hipDeviceSynchronize();

// copy.modified. d_data (GPU) to input_data (CPU) -> overwrite
            hipMemcpy(input_data.data(), d_data, sizeof(data_type) * input_data.size(), hipMemcpyDeviceToHost);
 
            auto fft_stop = std::chrono::high_resolution_clock::now();
//--------- end stop watch
//        std::cout<<"Time elapsed: "<<std::chrono::duration_cast<std::chrono::nanoseconds>(fft_stop - fft_start).count()<<",ns\n";

//        std::printf("Output array:\n");
            for (int i = 0; i < Nlauf; i++) {
                std::cout<<"cuFFTWout,0,0,"<<k<<","<<i<<","<<i<<","<<input_data[i].real()<<","<<input_data[i].imag()<<"\n";

            }
//        std::printf("=====\n");
//            std::cout<<"cufft,SINpND,"<<Nlauf<<",FORWARD,"<<k<<","<<std::chrono::duration_cast<std::chrono::nanoseconds>(fft_stop - fft_start).count()<<",ns\n";
        }


        /* free resources */
        hipFree(d_data);

        hipfftDestroy(plan);

        hipStreamDestroy(stream);

        hipDeviceReset();
    }
    return 0;
}
